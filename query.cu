#include <stdio.h>
/* Print info about connected Nvidia GPUs */
void query_GPUs() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Major revision number: %d\n", prop.major);
        printf("  Minor revision number: %d\n", prop.minor);
        printf(  "Total shared memory per block: %u\n",  prop.sharedMemPerBlock);
        printf("  Total registers per block: %d\n",  prop.regsPerBlock);
        printf(  "Warp size: %d\n",  prop.warpSize);
        printf("  Maximum memory pitch: %u\n",  prop.memPitch);
        printf("  Maximum threads per block: %d\n",  prop.maxThreadsPerBlock);
        printf("  Clock rate (KHz): %d\n",  prop.clockRate);
        printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Total VRAM (Bytes): %u\n",  prop.totalGlobalMem);
        printf("  Total constant memory (Bytes): %u\n",  pro.totalConstMem);
        printf("  Number of SMs: %d\n", prop.multiProcessorCount);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Kernel execution timeout: %s\n",  (prop.kernelExecTimeoutEnabled ? "Yes" : "No"));
        printf("  Concurrent copy and execution: %s\n",  (pro.deviceOverlap ? "Yes" : "No"));
        printf("  Concurrent kernels: %s\n",  (prop.concurrentKernels ? "Yes" : "No"));
    }

}

int main(int argc, char ** argv) {

    query_GPUs();

    return 0;
}
