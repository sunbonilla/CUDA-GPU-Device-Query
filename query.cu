#include <stdio.h>
/* Print info about connected Nvidia GPUs */
void query_GPUs() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
        printf("  Total VRAM (Bytes): %u\n",  prop.totalGlobalMem);
        printf("  Number of SMs: %d\n", prop.multiProcessorCount);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Kernel execution timeout: %s\n",  (prop.kernelExecTimeoutEnabled ? "Yes" : "No"));
    }

}

int main(int argc, char ** argv) {

    query_GPUs();

    return 0;
}
